#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"

#include <GL/freeglut.h>
#ifdef __APPLE__
    #include <GLUT/glut.h>
    #include <OpenGL/glext.h>
    #include <OpenGL/gl.h>
    #include <OpenGL/glu.h>
#else
    #include <GL/glut.h>
    #include <GL/glext.h>
    #include <GL/gl.h>
    #include <GL/glu.h>
#endif

// Size of window in pixels, both width and height
#define WINDOW_DIM            512

using namespace std;

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;
const int maxIt = 2000; // Msximum Iterations

int updateRate = 50;
//-------------host------------------
int pixels = WINDOW_DIM * WINDOW_DIM;
int window = WINDOW_DIM;
unsigned int w = WINDOW_DIM;
int* p = &window;

int num=0;
int* ifMBSet;
int* itera;
float* min_r = &(minC.r);
float* min_i = &(minC.i);
float* max_r = &(maxC.r);
float* max_i = &(maxC.i);

GLfloat* image=NULL;
GLfloat* image2=NULL;
//------------------------------------





// Define the RGB Class
class RGB
{
public:
    RGB()
        : r(0), g(0), b(0) {}
    RGB(double r0, double g0, double b0)
        : r(r0), g(g0), b(b0) {}
public:
    double r;
    double g;
    double b;
};

RGB* colors = 0; // Array of color values

void InitializeColors()
{
  colors = new RGB[maxIt + 1];
  for(int i = 0; i < 4; ++i)
  {
    colors[maxIt] = RGB(drand48(), drand48(), drand48());
  }
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 5)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); // black
}


//-------------------------- my code ---------------------------

__global__ void isMBpoint(float* min_r,float* min_i,float* max_r,float* max_i,int* window,int* ifMB,int* ite)
{
    Complex c(0.0,0.0);
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int id = x + y * blockDim.x * gridDim.x;
    int N = *window;

    if(id<N*N)
    {
      int px= id / N;
      int py= id % N;
      float minr = *min_r;
      float mini = *min_i;
      float maxr = *max_r;
      float maxi = *max_i;
      c.r = (minr) + (maxr - minr)*1.0/N * px;
      c.i = (mini) + (maxi - mini)*1.0/N * py;

      Complex Z(c);
      int result = 1;
      int i;
      for(i=1;i<=2000;i++)
      {
          Z=Z*Z+c;
          if (Z.magnitude2()>4)   //mag^2 > 4
          {
              result=0;
              break;
          }
          
      }

      ite[id]=i;
      ifMB[id]=result;
    }
}

void AddRange(float minr,float mini,float maxr, float maxi, int& num)
{
  Complex* tmp=(Complex*)malloc(sizeof(Complex)*(num+1));
  for(int i=0;i<num;i++)
  {
    new (tmp+i) Complex(dev_minC[i].r, dev_minC[i].i);
  }
  new (tmp+num) Complex(minr, mini);
  for(int i=0;i<num;i++)
  {
    dev_minC[i].~Complex();
  }
  free(dev_minC);
  dev_minC=tmp;

  tmp=(Complex*)malloc(sizeof(Complex)*(num+1));
  for(int i=0;i<num;i++)
  {
    new (tmp+i) Complex(dev_maxC[i].r, dev_maxC[i].i);
  }
  new (tmp+num) Complex(maxr, maxi);
  for(int i=0;i<num;i++)
  {
    dev_maxC[i].~Complex();
  }
  free(dev_maxC);
  dev_maxC=tmp;

  min_r = & dev_minC[num].r;
  min_i = & dev_minC[num].i;
  max_r = & dev_maxC[num].r;
  max_i = & dev_maxC[num].i;

  num++;
    
}

void RmRange(int& num)
{
  if(num>1)
  {
    Complex* tmp=(Complex*)malloc(sizeof(Complex)*(num-1));
    for(int i=0;i<num-1;i++)
    {
      new (tmp+i) Complex(dev_minC[i].r, dev_minC[i].i);
    }
    for(int i=0;i<num;i++)
    {
      dev_minC[i].~Complex();
    }
    free(dev_minC);
    dev_minC=tmp;

    tmp=(Complex*)malloc(sizeof(Complex)*(num-1));
    for(int i=0;i<num-1;i++)
    {
      new (tmp+i) Complex(dev_maxC[i].r, dev_maxC[i].i);
    }
    for(int i=0;i<num;i++)
    {
      dev_maxC[i].~Complex();
    }
    free(dev_maxC);
    dev_maxC=tmp;

    min_r = & dev_minC[num-2].r;
    min_i = & dev_minC[num-2].i;
    max_r = & dev_maxC[num-2].r;
    max_i = & dev_maxC[num-2].i;

    num--;
  }
}

void CudaCalcu()
{
    int* d_ifMBSet;
    int* d_itera;
    int* N;
    hipMalloc((void **)&N, sizeof(int));
    hipMalloc((void **)&d_ifMBSet, pixels * sizeof(int));
    hipMalloc((void **)&d_itera, pixels * sizeof(int));

    float* d_min_r;
    float* d_min_i;
    float* d_max_r;
    float* d_max_i;
    hipMalloc((void **)&d_min_r, sizeof(float));
    hipMalloc((void **)&d_min_i, sizeof(float));
    hipMalloc((void **)&d_max_r, sizeof(float));
    hipMalloc((void **)&d_max_i, sizeof(float));

    //----------mem copy-------------

    hipMemcpy(N, p, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_min_r, min_r, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_min_i, min_i, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_max_r, max_r, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_max_i, max_i, sizeof(float), hipMemcpyHostToDevice);
    
    dim3 blocks(64, 128);
    dim3 threads(4, 8);
    isMBpoint<<<blocks, threads>>>(d_min_r, d_min_i, d_max_r, d_max_i, N, d_ifMBSet, d_itera);

    hipMemcpy(ifMBSet, d_ifMBSet, pixels * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(itera, d_itera, pixels * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_ifMBSet);
    hipFree(d_itera);
    hipFree(N);
    hipFree(d_min_r);
    hipFree(d_min_i);
    hipFree(d_max_r);
    hipFree(d_max_i);
}
void CalImage()
{
  if(image==NULL)
  {
    image = new GLfloat[pixels*3];
  }
  for(int i=0;i<pixels;i++)
  {
    int px = i/window;
    int py = i%window;
    int id = (py)*window+(px);
    if(ifMBSet[i]==0)
    {
      image[id*3] = colors[itera[i]-1].r;
      image[id*3+1] = colors[itera[i]-1].g;
      image[id*3+2] = colors[itera[i]-1].b;
    }
    else
    {
      image[id*3] = 0.0;
      image[id*3+1] = 0.0;
      image[id*3+2] = 0.0;
    }
    
  }
  if(image2==NULL)
  {
    image2 = new GLfloat[pixels*3];
  }
  for(int i=0;i<pixels;i++)
  {
      image2[i*3]=image[i*3];
      image2[i*3+1]=image[i*3+1];
      image2[i*3+2]=image[i*3+2];

  }

}
//-------------------------------------------------------------
//-------------------imitation-------------------

void init()
{
  //select clearing (background) color
  glClearColor(0.0, 0.0, 0.0, 0.0);
  glShadeModel(GL_FLAT);
}


void reshape(int w, int h)
{
  glViewport(0,0, (GLsizei)w, (GLsizei)h);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0.0, (GLdouble)w, (GLdouble)0.0, h, (GLdouble)-w, (GLdouble)w);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
}


void timer(int)
{
  glutPostRedisplay();
  glutTimerFunc(1000.0 / updateRate, timer, 0);
}
//-------------------------------------------------
void showMandelbrot()
{
    glClear(GL_COLOR_BUFFER_BIT);
    glLoadIdentity();
    gluLookAt(0.0, 0.0, 5.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);

    glMatrixMode(GL_PROJECTION);
    gluOrtho2D(-1.0,1.0,-1.0,1.0);   //x1 x2    y1 y2

  glDrawPixels(w, w, GL_RGB, GL_FLOAT, image2);

  glutSwapBuffers();
}


void Keyboard(unsigned char key, int x, int y)
{
    if (key == 'b')
    {
      RmRange(num);
      CudaCalcu();
      cout<<num<<"#"<<endl<<endl<<endl;
      CalImage();
    }
}

int px1,py1,px2,py2;
float xx1,yy1,xx2,yy2;
int button_down=0;
void Mouse(int button, int state, int x, int y)
{
    if(button == GLUT_LEFT_BUTTON && state == GLUT_DOWN) 
    {
 //     cout<<"DOWN"<<x<<","<<y<<endl;
      px1=x;
      py1=y;
      button_down=1;
    }
    if(button == GLUT_LEFT_BUTTON && state == GLUT_UP) 
    {
 //     cout<<"UP"<<x<<","<<y<<endl;
      px2=x;
      py2=y;
      button_down=0;

   
      int widthx=0; 
      int widthy=0;
      int width=0;
      if(px1>px2)
      {
        widthx=px1-px2;
      }
      else{
        widthx=px2-px1;
      }
      if(py1<py2)
      {
        widthy=py2-py1;
      }
      else{
        widthy=py1-py2;
      }

      if(widthx<widthy){
        width=widthx;
      }
      else{
        width=widthy;
      }
      
      if(px1>px2)
      {
        px2=px1-width;  
        xx1=dev_minC[num-1].r + (dev_maxC[num-1].r-dev_minC[num-1].r)*(px2*1.0/window);
        xx2=dev_minC[num-1].r + (dev_maxC[num-1].r-dev_minC[num-1].r)*(px1*1.0/window);
      }
      else{
        px2=px1+width;
        xx1=dev_minC[num-1].r + (dev_maxC[num-1].r-dev_minC[num-1].r)*(px1*1.0/window);
        xx2=dev_minC[num-1].r + (dev_maxC[num-1].r-dev_minC[num-1].r)*(px2*1.0/window);
      }
      if(py1<py2)
      {
        py2=py1+width;
        yy1=dev_maxC[num-1].i - (dev_maxC[num-1].i-dev_minC[num-1].i)*(py2*1.0/window);
        yy2=dev_maxC[num-1].i - (dev_maxC[num-1].i-dev_minC[num-1].i)*(py1*1.0/window);

      }
      else{
        py2=py1-width;
        yy1=dev_maxC[num-1].i - (dev_maxC[num-1].i-dev_minC[num-1].i)*(py1*1.0/window);
        yy2=dev_maxC[num-1].i - (dev_maxC[num-1].i-dev_minC[num-1].i)*(py2*1.0/window);

      }

      AddRange(xx1, yy1, xx2, yy2,num);
      CudaCalcu();
  //    cout<<num<<"#"<<endl<<endl<<endl;
      CalImage();
      glutDisplayFunc(showMandelbrot);
    }
}

void OnMouseMove(int x, int y)          /*当鼠标移动时会回调该函数*/

{
    if(button_down)         /*如果鼠标没有按下则不改变摄像机位置*/

    {
      for(int i=0;i<pixels;i++)
      {
        image2[i*3]=image[i*3];
        image2[i*3+1]=image[i*3+1];
        image2[i*3+2]=image[i*3+2];
      }
      if(x<px1){
        for(int i=x;i<=px1;i++)
        {
          
          image2[((window-y)*window+i)*3]=0;
          image2[((window-y)*window+i)*3+1]=0;
          image2[((window-y)*window+i)*3+2]=1;
          image2[((window-y-1)*window+i)*3]=0;
          image2[((window-y-1)*window+i)*3+1]=0;
          image2[((window-y-1)*window+i)*3+2]=1;

          image2[((window-py1)*window+i)*3]=0;
          image2[((window-py1)*window+i)*3+1]=0;
          image2[((window-py1)*window+i)*3+2]=1;
          image2[((window-py1-1)*window+i)*3]=0;
          image2[((window-py1-1)*window+i)*3+1]=0;
          image2[((window-py1-1)*window+i)*3+2]=1;
         }

      }
      else
      {
          for(int i=px1;i<=x;i++)
        {
          
          image2[((window-y)*window+i)*3]=0;
          image2[((window-y)*window+i)*3+1]=0;
          image2[((window-y)*window+i)*3+2]=1;
          image2[((window-y-1)*window+i)*3]=0;
          image2[((window-y-1)*window+i)*3+1]=0;
          image2[((window-y-1)*window+i)*3+2]=1;

          image2[((window-py1)*window+i)*3]=0;
          image2[((window-py1)*window+i)*3+1]=0;
          image2[((window-py1)*window+i)*3+2]=1;
          image2[((window-py1-1)*window+i)*3]=0;
          image2[((window-py1-1)*window+i)*3+1]=0;
          image2[((window-py1-1)*window+i)*3+2]=1;
        }

      }

      if(y<py1){

        for(int i=y;i<=py1;i++){
          
          image2[((window-i)*window+x)*3]=0;
          image2[((window-i)*window+x)*3+1]=0;
          image2[((window-i)*window+x)*3+2]=1;
          image2[((window-i)*window+x+1)*3]=0;
          image2[((window-i)*window+x+1)*3+1]=0;
          image2[((window-i)*window+x+1)*3+2]=1;

          image2[((window-i)*window+px1)*3]=0;
          image2[((window-i)*window+px1)*3+1]=0;
          image2[((window-i)*window+px1)*3+2]=1;
          image2[((window-i)*window+px1+1)*3]=0;
          image2[((window-i)*window+px1+1)*3+1]=0;
          image2[((window-i)*window+px1+1)*3+2]=1;
        }
      }
      else
      {
        for(int i=py1;i<=y;i++){
          image2[((window-i)*window+x)*3]=0;
          image2[((window-i)*window+x)*3+1]=0;
          image2[((window-i)*window+x)*3+2]=1;
          image2[((window-i)*window+x+1)*3]=0;
          image2[((window-i)*window+x+1)*3+1]=0;
          image2[((window-i)*window+x+1)*3+2]=1;

          image2[((window-i)*window+px1)*3]=0;
          image2[((window-i)*window+px1)*3+1]=0;
          image2[((window-i)*window+px1)*3+2]=1;
          image2[((window-i)*window+px1+1)*3]=0;
          image2[((window-i)*window+px1+1)*3+1]=0;
          image2[((window-i)*window+px1+1)*3+2]=1;
        }
      }
    }

}

//--------------------------------------------------


int main(int argc, char** argv)
{
  InitializeColors();

  // Set up necessary host and device buffers
  for(int i=0;i<2001;i++)
  {
//    cout<<"("<<colors[i].r<<","<<colors[i].g<<","<<colors[i].b<<")"<<endl;
  }

  ifMBSet = new int[pixels];
  itera = new int[pixels];
  for(int i=0;i<pixels;i++)
  {
    ifMBSet[i] = 0;
    itera[i] = 0;
  }
 
  //---------------------------------------------------

  AddRange(minC.r, minC.i, maxC.r, maxC.i,num);
  CudaCalcu();
//  cout<<num<<"#"<<endl<<endl<<endl;
  CalImage();
  

  // Initialize OPENGL here
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(512, 512);
  glutInitWindowPosition(100, 100);
  glutCreateWindow("Mandelbrot");
  init();

  glutDisplayFunc(showMandelbrot);
  glutKeyboardFunc(Keyboard);
  glutMouseFunc(Mouse);  
  glutMotionFunc(OnMouseMove); 
 
  glutReshapeFunc(reshape);
  updateRate=10;
  glutTimerFunc(1000.0 / updateRate, timer, 0);

  glutMainLoop(); // THis will callback the display, keyboard and mouse


  delete [] ifMBSet;
  delete [] itera;
  delete [] colors;

  return 0;
   
}
